#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "wtime.h"

#include<iostream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<ostream>

#include <hip/hip_runtime_api.h>
#include "common.h"
#include "determinante_cuda.h"



extern void gpu_numbers (struct prop_devices *&gpus, int *devices)
{
	int i;
	hipGetDeviceCount(devices);
	gpus = (struct prop_devices *)malloc(sizeof(struct prop_devices)*(*devices));
	for (i = 0; i < *devices; i++) { gpus[i].id = i; gpus[i].busy = false; }
}


extern FLOAT  calculate_row(FLOAT *dataSetMatrix,FLOAT *d_A, FLOAT *d_A_r, FLOAT *d_output, hipsolverHandle_t solver_handle, hipStream_t stream, int *dev_info, dim3 bk, dim3 th, unsigned int sh, int STRIDE, int rows, int columns, int n_th, int i)
{

	//hipSetDevice(n_th);
	//...	
	//return (det*det);
}


FLOAT extract_number (char *buffer, int *pos, char separator);
extern FLOAT *read_data (char *input, int rows, int columns, FLOAT *m, char separator)
{
     int pos = 0,i = 0, j = 0;
     //printf("%d %d\n",rows,columns);
     //m = (FLOAT *)malloc(rows*columns*sizeof(FLOAT));
     hipHostMalloc(&m,rows*columns*sizeof(FLOAT));

     char buffer[MAX_LINE+1];// = (char *)malloc(sizeof(char)*MAX_LINE);
     FILE *fp;
     if ((fp =  fopen(input,"r")) == NULL) {
     	printf("ReadInput(): Can't open file \"%s\"\n", input);
	exit(1);
	}
     while((fgets(buffer,MAX_LINE,fp) != NULL) && (i < rows))
     {
	for (j = 0;j < columns;j++)
	{
		m[i*columns + j] = extract_number(buffer,&pos,separator);
		//printf("%f \n",m[i*columns + j]);
	}
	pos = 0;
	i++;
       //printf("%u %u\n",i,j);
     }
     fclose(fp);
     return (m);
}

FLOAT extract_number (char *buffer, int *pos, char separator)
{       int k = 0;
        char c;
        //c = buffer[*pos];
        char temp[100];
        //type_data number;
        while (buffer[*pos] == ' ') (*pos)++;
 	c = buffer[*pos];
	while (c != separator)
	{
		temp[k] = buffer[*pos];
		k++;(*pos)++;
	        c = buffer[*pos];
	}
	(*pos)++;
	temp[k]='\0';
	if (c == '\n') temp[k-1] = '\0';
	//printf("%s\n",temp);
	return ((FLOAT)atof(temp));
}



__global__ void matCov_gpu(FLOAT * dataSetMatrix, int rows, int columns, FLOAT *output, int loop)
{
  int i=threadIdx.x+blockDim.x*blockIdx.x;
  int N=columns*columns;
  //si se lanzan mas hilos que datos, que no trabajen
  if(i<N)
  {
    int a,b,c;
    FLOAT sum=0;
    //idea Baldo , como obtener la fila y columna de una matriz 1D.
    a = i/columns; // teniendo una matriz de 3x3, si el id es 7/3=2
    b = i%columns; // teniendo una matriz de 3x3, si el id es 7%3=1

    for ( c = 0; c < rows; c++ ) {
  	  sum += ( dataSetMatrix[c*columns+a] - dataSetMatrix[loop*columns+a] ) * ( dataSetMatrix[c*columns+b] - dataSetMatrix[loop*columns+b] );
    }
    //Guardamos con un hilo un elemento
    output[i] = sum / rows;  
  }
  
}

__global__ void scale(double *a, int size, int index) {
    int i;
    // Calculo de la posición de inicio para recorrer el vector 
    int start = (index*size + index);
    // Calculo de la posición final para recorrer el vector 
    int end = (index*size + size);
    // Ciclo for que recorre el vector desde la posición start+1 hasta end 
    for (i = start + 1; i<end; i++) {
        //dividimos cada elemento del vector desde la posición start+1 hasta end con el primer elemento del vector 
        a[i] = (a[i] / a[start]);
    }
}
__global__ void reduce(double *a, int size, int index) {
    int i;
    //indice del hilo actual 
    int tid = threadIdx.x;
    //Calculamos la posición de inicio para recorrer el vector 
    int start = ((index + tid + 1)*size + index);
    // Calculamos la posición final para recorrer el vector 
    int end = ((index + tid + 1)*size + size);
    // Ciclo for que recorre el arreglo desde la posición start+1 hasta end 
    for (i = start + 1; i<end; i++) {
        // restamos cada elemento del vector desde la posición start+1 hasta end con el primer elemento del vector multiplicado 
        //por otro elemento del vector en la posición (index*size) + (index + (i - start))
        a[i] = a[i] - (a[start] * a[(index*size) + (index + (i - start))]);
    }
}
//author --> mohrobati
//https://github.com/mohrobati/DeterminantCUDA/blob/main/parallel/fp/kernel.cu
void gaussianElimination(double* dev_a, int N) {
    int i;
    //for que recorre desde 0 hasta N 
    for (i = 0; i<N; i++) {
        // Ejecutamos la función scale con un solo bloque y un solo hilo 
        scale << <1, 1 >> >(dev_a, N, i);
        // Ejecutamos la función reduce con un solo bloque y (N - i - 1) hilos 
        reduce << <1, (N - i - 1) >> >(dev_a, N, i);
    }
}
FLOAT calculateDet(FLOAT *A, int N)
{
	//A esta localizado en el device
	//por eso necesitamos pasarlo al host
	FLOAT *h_a;
	FLOAT det=1.0;
	h_a=(FLOAT *)malloc(N*N * sizeof(FLOAT));
	 
	//gauss
	gaussianElimination(A, N);
	hipMemcpy(h_a, A, N*N * sizeof(FLOAT), hipMemcpyDeviceToHost);

	// Calculamos determinante como producto de los elementos de la diagonal
    for (int i = 0; i < N; i++){
		det *= h_a[i * N + i];
	}
	return det;

}


FLOAT Calculo_det_MAT (FLOAT *dataSetMatrix, int rows, int columns)
{
	FLOAT f1=0;
	FLOAT det = 0;
	int i;
	//CARGA DE POSIBLES DATOS
  hipSetDevice(0);
  	size_t size=rows * columns * sizeof( FLOAT );
	FLOAT *d_dataSetMatrix;
	hipMalloc((void **)&d_dataSetMatrix,size);
	hipMemcpy(d_dataSetMatrix,dataSetMatrix,size,hipMemcpyHostToDevice);
	 // Allocate device memory for the output matrix
	 FLOAT *output;
	 output=(FLOAT*)malloc(columns*columns*sizeof(FLOAT));
	FLOAT *d_output;
	hipMalloc((void **)&d_output, columns*columns*sizeof(FLOAT));
 

  int NUM_THREADS=128;
   // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = ((columns*columns) + NUM_THREADS - 1) / NUM_THREADS;
  dim3 hilos(NUM_THREADS);
  //dim3 bloques(((columns*columns)/NUM_THREADS)+1);
  dim3 bloques(NUM_BLOCKS);
  
	//PROCESAMIENTO POR FILA
	for ( i = 0; i < rows; i++ ) 
	{

		//se elegio una matriz bidimensional para simular los dos primeros fors del codigo secuencial
		  matCov_gpu<<<bloques, hilos>>>(d_dataSetMatrix, rows, columns, d_output, i);
    //printf("%d\n",i);
		//hipDeviceSynchronize();
		det=calculateDet(d_output,columns);
		f1 = f1 + 1 / sqrt( det );
    }
return f1;

}

